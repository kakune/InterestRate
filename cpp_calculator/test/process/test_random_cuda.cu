#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <vector>

#include "process/random.hpp"

__global__ void myKernel( Process::Random::StdBrownPlain* obj,
                          double* inpResult )
{
    obj->dGenerateRandomVal( inpResult );
}

int main( int argc, char* argv[] )
{
    std::size_t lNTerms = 5;
    std::size_t lNPath  = 30;
    double dt           = 0.1;
    std::vector<double> lTerms( lNTerms, 0 );
    for ( std::size_t iTerm = 1; iTerm < lNTerms; ++iTerm )
    {
        lTerms[iTerm] = lTerms[iTerm - 1] + dt;
    }
    auto lsTerms = std::make_shared<std::vector<double>>( lTerms );
    Process::Random::StdBrownPlain lPathObj( lNPath, lsTerms );
    Process::Random::StdBrownPlain* lpdPathObj;
    hipMalloc( &lpdPathObj, sizeof( Process::Random::StdBrownPlain ) );
    hipMemcpy( lpdPathObj, &lPathObj, sizeof( Process::Random::StdBrownPlain ),
                hipMemcpyHostToDevice );

    double* ldResult;
    hipMalloc( &ldResult, sizeof( double ) );

    myKernel<<<1, 1>>>( lpdPathObj, ldResult );
    hipDeviceSynchronize();
    double lResult;
    hipMemcpy( &lResult, ldResult, sizeof( double ), hipMemcpyDeviceToHost );
    std::cout << lResult << std::endl;

    // // デバイスからホストへ更新されたオブジェクトをコピー
    // hipMemcpy( &h_obj, d_obj, sizeof( MyDeviceClass ),
    //             hipMemcpyDeviceToHost );

    // // リソースを解放
    // hipFree( d_obj );

    return 0;
}